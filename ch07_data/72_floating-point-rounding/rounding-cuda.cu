
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void add(const float *a, const float *b,
                    float *result_ne, float *result_up,
                    float *result_down, float *result_zero) {
  int i = threadIdx.x;
  result_ne[i] = __fadd_rn(a[i], b[i]);
  result_up[i] = __fadd_ru(a[i], b[i]);
  result_down[i] = __fadd_rd(a[i], b[i]);
  result_zero[i] = __fadd_rz(a[i], b[i]);
}

void checkError(hipError_t err, const char *name) {
  if (err != hipSuccess) {
    fprintf(stderr, "%s: %s\n", name, hipGetErrorString(err));
    exit(1);
  }
}

int main() {
  size_t size = sizeof(float);
  float a = 0x1p0f;
  float b = 0x1p-100f;
  float result_ne, result_up, result_down, result_zero;

  hipError_t err;
  float *d_a, *d_b, *d_result_ne, *d_result_up, *d_result_down, *d_result_zero;
  err = hipMalloc(&d_a, size);
  checkError(err, "hipMalloc");
  err = hipMalloc(&d_b, size);
  checkError(err, "hipMalloc");
  err = hipMalloc(&d_result_ne, size);
  checkError(err, "hipMalloc");
  err = hipMalloc(&d_result_up, size);
  checkError(err, "hipMalloc");
  err = hipMalloc(&d_result_down, size);
  checkError(err, "hipMalloc");
  err = hipMalloc(&d_result_zero, size);
  checkError(err, "hipMalloc");

  err = hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
  checkError(err, "hipMemcpy");
  err = hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);
  checkError(err, "hipMemcpy");

  add<<<1, 1>>>(d_a, d_b, d_result_ne, d_result_up, d_result_down,
                d_result_zero);

  err = hipMemcpy(&result_ne, d_result_ne, size, hipMemcpyDeviceToHost);
  checkError(err, "hipMemcpy");
  err = hipMemcpy(&result_up, d_result_up, size, hipMemcpyDeviceToHost);
  checkError(err, "hipMemcpy");
  err = hipMemcpy(&result_down, d_result_down, size, hipMemcpyDeviceToHost);
  checkError(err, "hipMemcpy");
  err = hipMemcpy(&result_zero, d_result_zero, size, hipMemcpyDeviceToHost);
  checkError(err, "hipMemcpy");

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_result_ne);
  hipFree(d_result_up);
  hipFree(d_result_down);
  hipFree(d_result_zero);

  printf("nearesteven: %a\n", result_ne);
  printf("up: %a\n", result_up);
  printf("down: %a\n", result_down);
  printf("zero: %a\n", result_zero);
}
